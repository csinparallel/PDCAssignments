#include "hip/hip_runtime.h"
/** runTimeTrials.cpp uses MatrixTimer to time different Matrix operations.
 * Joel C. Adams, for CS 112 at Calvin University
 * Converted from C++ to C by Steven McKelvey, Calvin University
 * Adapted for CSinParallel.org.
 */

#include "MatrixTimer.cuh"
#include "Matrix.cuh"
#include <unistd.h>

char *buildLogFileName(unsigned numTrials);

int main()
{
	unsigned numTrials = 10; // number of trials to average
	char *logFileName = buildLogFileName(numTrials);

	MatrixTimer *matrixTimer = (MatrixTimer *)malloc(sizeof(MatrixTimer));
	initializeMatrixTimer(matrixTimer, numTrials, logFileName);
	matrixTimerRun(matrixTimer);

	freeMatrixTimer(matrixTimer); // free the members of matrixTimer
	free(matrixTimer);
	free(logFileName);
}

/** utility function to build unique log-file names
 * @param: numTrials, the number of trials to be performed
 * @return: a unique file name incorporating numTrials (returnString)
 */
char *buildLogFileName(unsigned numTrials)
{
	char emptyString[1] = "";
	char buf[16];
	sprintf(buf, "%02d", numTrials);
	char prefix[24];
	sprintf(prefix, "%strials-%s", emptyString, buf);
	char *result = (char *)malloc(sizeof(char) * 40);
	FILE *fin;
	unsigned count = 0;
	while (true)
	{
		++count;
		sprintf(buf, "-%03d.txt", count);
		sprintf(result, "%s%s", prefix, buf);
		if (access(result, F_OK) == -1)
		{ // if the file (log) does not already exist
			// create a new logfile for this set of trials
			fin = fopen(result, "w");
			break;
		}
		// otherwise, keep looping until you get to the next available log file name
	}
	fclose(fin);
	return result;
}