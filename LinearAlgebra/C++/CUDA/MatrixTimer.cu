#include "hip/hip_runtime.h"
/** MatrixTimer.cpp defines operations that can be performed on instances of MatrixTimer.
 * Joel Adams, for CS 112 at Calvin University.
 * Copyright Joel C. Adams, 2009.  All Rights Reserved.
 *
 * Adapted for CSinParallel.org by Steven McKelvey, Calvin University
 * Extended to include timing for matrix multiplication and equality
 */

#include "MatrixTimer.cuh"

/** explicit-value constructor
 * @param: numTrials, the number of time-trials for each operation
 *             logFile, a file where time-trial data are logged.
 * @postcondition: my instance variables have all been initialized.
 */

MatrixTimer::MatrixTimer(unsigned numTrials, const string &logFile)
{
    myNumTrials = numTrials;
    myTrialTimes = new double[numTrials];
    if (logFile != "")
    {
        myLogStream.open(logFile.c_str());
    }
    cout << "\nLoading matrices..." << flush;
    // the datatype of the values stored in these testFiles must be the same as the Item type in MatrixTimer.h
    myMatrix.fillText("testFiles/1024x1024.txt");
    myMatrix2.fillText("testFiles/1024x1024.txt");
}

/** destructor
 * @postcondition: my dynamic array has been returned to the heap &&
 *                 my log file has been closed.
 */
MatrixTimer::~MatrixTimer()
{
    delete[] myTrialTimes;
    if (myLogStream.is_open())
    {
        myLogStream.close();
    }
}

/** method to run time-trials
 * @postcondition: my time-trials have been run.
 */
void MatrixTimer::run()
{
    cout << endl;
    timeAddition();
    timeTranspose();
    timeEquality();
    timeMultiplication();
    cout << endl;
}

/** time the addition operation
 * @postcondition: the addition operation has been performed
 *                      myNumTrials times, &&
 *                 myLogStream contains the time-trial times and
 *                      their average &&
 *                 an asterisk has been display on the screen
 *                      for each time-trial &&
 *                 their average has been displayed on the screen.
 */
void MatrixTimer::timeAddition()
{
    cout << "Timing Matrix addition: \t" << flush;
    myLogStream << fixed << "\nTiming Matrix addition ...\n";
    double start = 0, stop = 0;
    for (unsigned i = 0; i < myNumTrials; ++i)
    {
        cout << " * " << flush;
        start = omp_get_wtime();
        myMatrix + myMatrix2;
        stop = omp_get_wtime();
        myTrialTimes[i] = stop - start;
        myLogStream << "\nTrial " << i << ": " << myTrialTimes[i];
    }
    double avgTime = average();
    cout << '\t' << fixed << setw(7) << setprecision(5) << avgTime << endl;
    myLogStream << "\nAverage: " << fixed << setw(7) << setprecision(5)
                << avgTime << endl;
}

/** utility method to average the time-trials in myTrialTimes.
 * @precondition: myNumTrials > 0 &&
 *                myTrialTimes contains that many trial-times
 * @return: the average of the trial-times in myTrialTimes.
 */
double MatrixTimer::average() const
{
    double sum = 0;
    for (unsigned i = 0; i < myNumTrials; ++i)
    {
        sum += myTrialTimes[i];
    }
    return sum / myNumTrials;
}

/** time the transpose operation
 * @postcondition: the transpose operation has been performed
 *                      myNumTrials times, &&
 *                 myLogStream contains the time-trial times and
 *                      their average &&
 *                 an asterisk has been display on the screen
 *                      for each time-trial &&
 *                 their average has been displayed on the screen.
 */
void MatrixTimer::timeTranspose()
{
    cout << "Timing Matrix transpose: \t" << flush;
    myLogStream << "\nTiming Matrix transpose ...\n";
    double start = 0, stop = 0;
    for (unsigned i = 0; i < myNumTrials; ++i)
    {
        cout << " * " << flush;
        start = omp_get_wtime();
        myMatrix.transpose();
        stop = omp_get_wtime();
        myTrialTimes[i] = stop - start;
        myLogStream << "\nTrial " << i << ": " << myTrialTimes[i];
    }
    double avgTime = average();
    cout << '\t' << fixed << setw(7) << setprecision(5) << avgTime << endl;
    myLogStream << "\nAverage: " << fixed << setw(7) << setprecision(5)
                << avgTime << endl;
}

/** time the boolean equality operation
 * @postcondition: the boolean equality operation has been performed
 *                      myNumTrials times, &&
 *                 myLogStream contains the time-trial times and
 *                      their average &&
 *                 an asterisk has been display on the screen
 *                      for each time-trial &&
 *                 their average has been displayed on the screen.
 */
void MatrixTimer::timeEquality()
{
    cout << "Timing Matrix equality: \t" << flush;
    myLogStream << fixed << "\nTiming Matrix equality ...\n";
    double start = 0, stop = 0;
    for (unsigned i = 0; i < myNumTrials; ++i)
    {
        cout << " * " << flush;
        start = omp_get_wtime();
        myMatrix == myMatrix2;
        stop = omp_get_wtime();
        myTrialTimes[i] = stop - start;
        myLogStream << "\nTrial " << i << ": " << myTrialTimes[i];
    }
    double avgTime = average();
    cout << '\t' << fixed << setw(7) << setprecision(5) << avgTime << endl;
    myLogStream << "\nAverage: " << fixed << setw(7) << setprecision(5)
                << avgTime << endl;
}

/** time the multiplication operation
 * @postcondition: the multiplication operation has been performed
 *                      myNumTrials times, &&
 *                 myLogStream contains the time-trial times and
 *                      their average &&
 *                 an asterisk has been display on the screen
 *                      for each time-trial &&
 *                 their average has been displayed on the screen.
 */
void MatrixTimer::timeMultiplication()
{
    cout << "Timing Matrix multiplication: \t" << flush;
    myLogStream << fixed << "\nTiming Matrix multiplication ...\n";
    double start = 0, stop = 0;
    for (unsigned i = 0; i < myNumTrials; ++i)
    {
        cout << " * " << flush;
        start = omp_get_wtime();
        myMatrix *myMatrix2;
        stop = omp_get_wtime();
        myTrialTimes[i] = stop - start;
        myLogStream << "\nTrial " << i << ": " << myTrialTimes[i];
    }
    double avgTime = average();
    cout << '\t' << fixed << setw(7) << setprecision(5) << avgTime << endl;
    myLogStream << "\nAverage: " << fixed << setw(7) << setprecision(5)
                << avgTime << endl;
}
